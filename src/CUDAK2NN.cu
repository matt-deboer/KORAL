#include "hip/hip_runtime.h"
/*******************************************************************
*   CUDAK2NN.cu
*   CUDAK2NN
*
*	Author: Kareem Omar
*	kareem.omar@uah.edu
*	https://github.com/komrad36
*
*	Last updated Oct 12, 2016
*******************************************************************/
//
// Fastest GPU implementation of a brute-force
// matcher for 512-bit binary descriptors
// in 2NN mode, i.e., a match is returned if the best
// match between a query vector and a training vector
// is more than a certain threshold number of bits
// better than the second-best match.
//
// Yes, that means the DIFFERENCE in popcounts is used
// for thresholding, NOT the ratio. This is the CORRECT
// approach for binary descriptors.
//
// This laboriously crafted kernel is EXTREMELY fast.
// 63 BILLION comparisons per second on a stock GTX1080,
// enough to match nearly 46,000 descriptors per frame at 30 fps (!)
//
// A key insight responsible for much of the performance of
// this insanely fast CUDA kernel is due to
// Christopher Parker (https://github.com/csp256), to whom
// I am extremely grateful.
//
// CUDA CC 3.0 or higher is required.
//
// All functionality is contained in the files CUDAK2NN.h
// and CUDAK2NN.cu. 'main.cpp' is simply a sample test harness
// with example usage and performance testing.
//

#include "koral/CUDAK2NN.h"
#include <stdio.h>

__global__ void
#ifndef __INTELLISENSE__
__launch_bounds__(256, 0)
#endif
CUDAK2NN_kernel(const hipTextureObject_t tex_q, const int num_q, const uint64_t* __restrict__ g_training, const int num_t, int* const __restrict__ g_match, const uint8_t threshold) {
	uint64_t train = *(g_training += threadIdx.x & 7);
	g_training += 8;
	uint64_t q[8];
	for (int i = 0, offset = ((threadIdx.x & 24) << 3) + (threadIdx.x & 7) + (blockIdx.x << 11) + (threadIdx.y << 8); i < 8; ++i, offset += 8) {
		const uint2 buf = tex1Dfetch<uint2>(tex_q, offset);
		asm("mov.b64 %0, {%1,%2};" : "=l"(q[i]) : "r"(buf.x), "r"(buf.y)); // some assembly required
	}	
	int best_i, best_v = 100000, second_v = 200000;
#pragma unroll 6
	for (int t = 0; t < num_t; ++t, g_training += 8) {
		uint32_t dist[4];
		for (int i = 0; i < 4; ++i) dist[i] = __byte_perm(__popcll(q[i] ^ train), __popcll(q[i + 4] ^ train), 0x5410);
		for (int i = 0; i < 4; ++i) dist[i] += __shfl_xor(dist[i], 1);
		train = *g_training;
		if (threadIdx.x & 1) dist[0] = dist[1];
		if (threadIdx.x & 1) dist[2] = dist[3];
		dist[0] += __shfl_xor(dist[0], 2);
		dist[2] += __shfl_xor(dist[2], 2);
		if (threadIdx.x & 2) dist[0] = dist[2];
		dist[0] = __byte_perm(dist[0] + __shfl_xor(dist[0], 4), 0, threadIdx.x & 4 ? 0x5432 : 0x5410);
		second_v = min(dist[0], second_v);
		if (dist[0] < best_v) {
			second_v = best_v;
			best_i = t;
			best_v = dist[0];
		}
	}
	const int idx = (blockIdx.x << 8) + (threadIdx.y << 5) + threadIdx.x;
	if (idx < num_q) g_match[idx] = second_v - best_v > threshold ? best_i : -1;
}

void CUDAK2NN(const void* const __restrict d_t, const int num_t, const hipTextureObject_t tex_q, const int num_q, int* const __restrict d_m, const int threshold) {
	CUDAK2NN_kernel<<<((num_q - 1) >> 8) + 1, { 32, 8 }>>>(tex_q, num_q, reinterpret_cast<const uint64_t*>(d_t), num_t, d_m, threshold);
	hipDeviceSynchronize();
}
